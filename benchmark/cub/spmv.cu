#define HIPCUB_STDERR

#include <cub/device/device_spmv.cuh>
#include <cub/util_allocator.cuh>
#include <cub/util_debug.cuh>

#include "api/types.h"

void spmv(int trans, const csr_desc<int, double> h_csr_desc, const csr_desc<int, double> d_csr_desc, const double *x,
          double *y) {
  double *d_values = const_cast<double *>(d_csr_desc.values);
  int *d_row_ptr = const_cast<int *>(d_csr_desc.row_ptr);
  int *d_col_index = const_cast<int *>(d_csr_desc.col_index);
  double *d_x = const_cast<double *>(x);
  double *d_y = const_cast<double *>(y);
  int rows = h_csr_desc.rows;
  int cols = h_csr_desc.cols;
  int nnz = h_csr_desc.nnz;
  // Caching allocator for device memory
  hipcub::CachingDeviceAllocator g_allocator(true);
  void *d_buffer = NULL;
  size_t d_buffer_size = 0;
  // Get buffer size
  CubDebugExit(hipcub::DeviceSpmv::CsrMV<double>(d_buffer, d_buffer_size, d_values, d_row_ptr, d_col_index, d_x, d_y, rows,
                                              cols, nnz, (hipStream_t)0, false));
  // Allocate an external buffer
  CubDebugExit(g_allocator.DeviceAllocate(&d_buffer, d_buffer_size));
  // Execute SpMV
  CubDebugExit(hipcub::DeviceSpmv::CsrMV<double>(d_buffer, d_buffer_size, d_values, d_row_ptr, d_col_index, d_x, d_y, rows,
                                              cols, nnz, (hipStream_t)0, false));
}